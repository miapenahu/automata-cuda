#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <unistd.h>
#include <stdbool.h> 
#include <SDL2/SDL.h>
//#include "SDL_ttf.h"
#include "/usr/include/SDL2/SDL_ttf.h"
//CUDA imports
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hiprand.h>


#include "util.h"
#include "logic.h"
#include "render.h"

#define SECONDS_TO_MICROSECONDS 1000000

//*calculate frames per second
struct timeval tval_before, tval_after, tval_result;
long int frame_count = 0;


void print_usage()
{
    printf("Usage: ./automata AUTOMATA\n");
    printf("     Langton's ant           -> langton\n");
    printf("     Conway's Game of Life   -> gameoflife\n");
    printf("     Falling Sand Simulator  -> sandsim\n");
}


int main(int argc, char **argv)
{
    int automata;//! automata defines the simulation we are running (see logic.h)
    char running_title[64] = {'\0'};
    char paused_title[64] = {'\0'};

    //Funciones de util.c
    startUtilTimers();
    time_t t;
    srand((unsigned int) time(&t));
    //initUtilFonts();

    if (argc < 2) {
        print_usage();
        return EXIT_FAILURE;
    } else if (strcmp(argv[1], "langton") == 0) {
        automata = LANGTONS_ANT;
        strncat(running_title, "LANGTONS_ANT", 48);
        strncat(paused_title, "LANGTONS_ANT", 48);
    } else if (strcmp(argv[1], "gameoflife") == 0) {
        automata = GAME_OF_LIFE;
        strncat(running_title, "THE GAME OF LIFE", 48);
        strncat(paused_title, "THE GAME OF LIFE", 48);
    } else if (strcmp(argv[1], "sandsim") == 0) {
        automata = FALLING_SAND_SIM;
        strncat(running_title, "FALLING SAND SIMULATOR", 48);
        strncat(paused_title, "FALLING SAND SIMULATOR", 48);
    } else {
        fprintf(stderr, "No such automata.\n");
        print_usage();
        return EXIT_FAILURE;
    }

    strncat(running_title, " - RUNNING", 48);
    strncat(paused_title, " - PAUSED", 48);

    if (SDL_Init(SDL_INIT_VIDEO) != 0) {
        fprintf(stderr, "SDL_INIT Error: %s\n", SDL_GetError());
        return EXIT_FAILURE;
    }

    //inicia ttf para los mensajes
    if(TTF_Init()==-1) {
        printf("TTF_Init: %s\n", TTF_GetError());
        exit(2);
    }

    	
    // load font.ttf at size 16 into font
    TTF_Font *font;
    font=TTF_OpenFont("/usr/share/fonts/truetype/ubuntu/Ubuntu-M.ttf", 16);
    if(!font) {
        printf("TTF_OpenFont: %s\n", TTF_GetError());
        // handle error
    }

    SDL_Window *window = SDL_CreateWindow(running_title, SDL_WINDOWPOS_UNDEFINED,
                                                                                SDL_WINDOWPOS_UNDEFINED, SCREEN_WIDTH,
                                                                                SCREEN_HEIGHT, SDL_WINDOW_SHOWN);

    if (window == NULL) {
        fprintf(stderr, "SDL_CreateWindow Error: %s\n", SDL_GetError());
        return EXIT_FAILURE;
    }

    SDL_Renderer *renderer = SDL_CreateRenderer(
            window, -1, SDL_RENDERER_ACCELERATED | SDL_RENDERER_PRESENTVSYNC);

    if (renderer == NULL) {
        SDL_DestroyWindow(window);
        fprintf(stderr, "SDL_CreateRenderer Error: %s\n", SDL_GetError());
        return EXIT_FAILURE;
    }

    state_t state = {.mode = RUNNING_MODE};

     //! INIT BOARD AWITCHES THE GAME ACORDING TO THE AUTOMATA VARIABLES THAT WAS SELECTRED AT THE START
    // INIT BOARD
    switch (automata) {
        case LANGTONS_ANT:{
            state.ant.x = N / 2;
            state.ant.y = N / 2;
            state.ant.dir = LEFT;
            break;
        }
        case GAME_OF_LIFE:{
            for (int x = 0; x < N; x++)
                for (int y = 0; y < N; y++){
                    int idx = (y * N) + x;
                    state.board[idx] = BLACK;
                }

            // GLIDER
            int idx = ((N / 2) * N) + (N / 2);
            state.board[idx] = WHITE;
            idx = (((N / 2) + 1) * N) + (N / 2);
            state.board[idx] = WHITE;
            idx = (((N / 2) + 2) * N) + (N / 2);
            state.board[idx] = WHITE;
            idx = (((N / 2) + 1) * N) + ((N / 2) - 2);
            state.board[idx] = WHITE;
            idx = (((N / 2) + 2) * N) + ((N / 2) - 1);
            state.board[idx] = WHITE;
            break;
        }
         //* initial state of the world
        case FALLING_SAND_SIM:{
            for (int x = 0; x < N; x++){
                for (int y = 0; y < N; y++){
                    int idx = (y * N) + x;
                    state.board[idx] = AIR;
                    
                    //make rock is the sum is small
                    if (y > (N-(N/2.1))) { //make the sea
                        state.board[idx] = WATER;
                    } else if (y > (N-(N/1.05))) { //make sand
                        state.board[idx] = SAND;
                    } 
                    
                    if(y < 40){
                        state.board[idx] = AIR;
                    }

                    /*
                    if (y < (N-(N*0.75))&& y > (N-(N*0.8)) && x < (N-(N*0.25)) && x > (N-(N*0.3) )) { //make sand
                        state.board[x][y] = ESTATICO;
                    } */
                }
            } 

            /*        
            // print the matrix state,board in the console to test
            for (int x = 0; x < N; x++){
                for (int y = 0; y < N; y++){
                    //printf("%d", state.board[x][y]);
                }
                //printf("\n");
            }
            */
          break;
        }
        default:{
            for (int x = 0; x < N; x++)
                for (int y = 0; y < N; y++){
                    int idx = (y * N) + x;
                    state.board[idx] = AIR;
                }
            break;
        }
    }


     //! event infinite loop, to switch events
    // is a event is fired it executes the corresponding function
    SDL_Event event;
    bool draw;
    int drawing_element = FIRE;
    int brushSize = 2;

    char dest[200]= "Fire";

    //============= CUDA INITIALIZATION ===============//

    // set up data size of board
    int nElem = N * N;
    size_t nBytesBoards = nElem * sizeof(u_int8_t);
    size_t nBytesBool = nElem * sizeof(bool);
    u_int8_t *d_board;
    //Calcular los tamaños de los arreglos
    size_t nBytesStates = nElem * sizeof(hiprandState);
    // malloc device global memory
    hipMalloc((u_int8_t **)&d_board, nBytesBoards);
    //Random functions in device
    hiprandState *d_random;
    //Seed to send to device
    unsigned int seed = (unsigned int) time(&t);
    // malloc random numbers in device
    hipMalloc((void**)&d_random, nBytesStates);
    // transfer data from host to device
    hipMemcpy(d_board, state.board, nBytesBoards, hipMemcpyHostToDevice);

    //=================================================//

    while (state.mode != QUIT_MODE) {
      //! while loop to search for events and handle them doing an action dependong on the game
        while (SDL_PollEvent(&event)) {
            switch (event.type) {
                
                case SDL_QUIT:{
                    state.mode = QUIT_MODE;
                    break;
                }
                case SDL_MOUSEBUTTONDOWN:{

                  draw= true;

                    if (automata!=FALLING_SAND_SIM)
                    {
                      state.mode = PAUSED_MODE;
                      SDL_SetWindowTitle(window, paused_title);
                    }
                
                    int x = event.button.x / CELL_WIDTH;
                    int y = event.button.y / CELL_HEIGHT;
                    int idx = (y * N) + x;

                    // TOGGLES BETWEEN EACH ELEMENT TYPE WITH EACH CLICK
                    switch (automata) {
                        case GAME_OF_LIFE:{
                            state.board[idx] = (state.board[idx] + 1) % 2;
                            break;
                        }
                        // USE MODULE 9 TO ONLY GET A NUMBER BETWEEN 0 AND 9 THAT ARE HE NUMBER OF COLORS
                        case FALLING_SAND_SIM:{
                        // alter teh state of pixel with each click
                          if (draw)
                          {
                            int mouseix = event.motion.x;
                            int mouseiy = event.motion.y;
                            int mousex = mouseix / CELL_WIDTH;
                            int mousey = mouseiy / CELL_HEIGHT;
                           
                             for(int y = max(0,mousey-brushSize); y < min(N-1, mousey+brushSize); ++y){
                              for(int x = max(0,mousex-brushSize); x < min(N-1, mousex+brushSize); ++x){
                                int idx = (y * N) + x;
                                state.board[idx] = drawing_element;
                              }
                           }}
                            
                        break;
                        }
                    }
                  break;
                }

                // if the click is pressed and there is movement the mouse will draw any picture
                case SDL_MOUSEMOTION:{
                    if (draw)
                    {
                        int mouseix = event.motion.x;
                        int mouseiy = event.motion.y;
                        int mousex = mouseix / CELL_WIDTH;
                        int mousey = mouseiy / CELL_HEIGHT;
                        
                        
                        for(int y = max(0,mousey-brushSize); y < min(N-1, mousey+brushSize); ++y){
                              for(int x = max(0,mousex-brushSize); x < min(N-1, mousex+brushSize); ++x){
                                 int idx = (y * N) + x;
                                 state.board[idx] = drawing_element;
                              }
                            }

                    }
                    break;
                }
                case SDL_MOUSEBUTTONUP:{
                    draw=false;
                    break;
                }
                //event if left is used
                case SDLK_LEFT:{
                        //do something with left arrow
                    break;
                }
                case SDL_KEYDOWN:{
                    if (event.key.keysym.sym == ' ') {
                        state.mode = RUNNING_MODE + PAUSED_MODE - state.mode;
                        SDL_SetWindowTitle(window, state.mode ? paused_title : running_title);
                    }  //*makes a meteorite effect if m or M are pressed 
                    else if (event.key.keysym.sym == 'm' || event.key.keysym.sym == 'M') {
                      int random = rand() % 19;
                        for (int x = N/20*random; x < N/20*random + N/20; x++)
                            for (int y = 0; y < N/20; y++){
                                int idx = (y * N) + x;
                                state.board[idx] = (rand() % 2) ? ROCK : FIRE;
                            }
                    }           
                    else if (event.key.keysym.sym == 'f' || event.key.keysym.sym == 'F') {
                      drawing_element = FIRE;
                      
                      char aa[]= "Fire";
                      strcpy(dest,aa);
                 
                    }                                       
                    else if (event.key.keysym.sym == 's' || event.key.keysym.sym == 'S') {
                      drawing_element = SAND;
                      char bb[]= "Sand";
                      strcpy(dest,bb);
                    }  
                    else if (event.key.keysym.sym == 'a' || event.key.keysym.sym == 'A') {
                      drawing_element = AIR;
                      char cc[]= "Air";
                      strcpy(dest,cc);
                    }                              
                    else if (event.key.keysym.sym == 'r' || event.key.keysym.sym == 'R') {
                      drawing_element = ROCK;
                      char dd[]= "Rock";
                      strcpy(dest,dd);
                    }
                    else if (event.key.keysym.sym == 'w' || event.key.keysym.sym == 'W') {
                      drawing_element = WATER; 
                      char ee[]= "Water";
                      strcpy(dest,ee);
                  } 

                    else if (event.key.keysym.sym == 'h' || event.key.keysym.sym == 'H') {
                      drawing_element = HUMO;
                      char ff[]= "Humo";
                      strcpy(dest,ff);
                   } 
                    else if (event.key.keysym.sym == 'e' || event.key.keysym.sym == 'E') {
                      drawing_element = ESTATICO; 
                      char gg[]= "Estatico";
                      strcpy(dest,gg);} 
                    else if (event.key.keysym.sym == 'o' || event.key.keysym.sym == 'O') {
                      drawing_element = OIL;
                      char hh[]= "Oil";
                      strcpy(dest,hh); } 
                    else if (event.key.keysym.sym == '+' ) {
                      //print brush size
                      if (brushSize<N/5)
                      {
                       brushSize =  brushSize + 1;
                      }
                      
                    }
                    else if (event.key.keysym.sym == '-') {
                      if (brushSize>1)
                      {
                       brushSize =  brushSize - 1;
                      }
                       }
                      
                break;
                }
            }
        }

        SDL_SetRenderDrawColor(renderer, 255, 255, 255, 255);
        SDL_RenderClear(renderer);

        render_grid(renderer, &state);

        if (state.mode == RUNNING_MODE)
            //usleep((1.0 / MOVES_PER_SECOND) * SECONDS_TO_MICROSECONDS);


        switch (automata) {
            case LANGTONS_ANT:{
                langtons_ant(renderer, &state);
                break;
            }
            case GAME_OF_LIFE:{
                game_of_life(renderer, &state);
                break;
            }
            case FALLING_SAND_SIM:{
                world_sand_simOnGPU(renderer, &state, d_board, d_random, seed);
                break;
            }
        }

        //Función para imprimir un texto indicando la fuente, posición y color de la fuente
        //renderText(SDL_Renderer *renderer, TTF_Font *font, int r, int g, int b, char stringText[], int x, int y){
        renderText(renderer,font, 6, 150, 78, dest, 0, 0);
        char str[32];
        sprintf(str, "BrushSize: %d", brushSize);  
        renderFormattedText(renderer,str,100, 0);
        SDL_RenderPresent(renderer);

    }

    SDL_DestroyWindow(window);
    TTF_Quit();
    // you could SDL_Quit(); here...or not.
    SDL_Quit();

    // free device global memory
    hipFree(d_board);
    hipFree(d_random);

    return EXIT_SUCCESS;
}
